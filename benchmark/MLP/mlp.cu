#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <sys/time.h>

#include <fstream>
#include <iostream>
#include <string>

#include "apis_cu.h"
#include "hip/hip_runtime.h"
#include ""
#define BLOCK_DIM 10

__global__ void matrix_mul_gpu(int64_t *M, int64_t *N, int64_t *P, int64_t widthA, int64_t heightA,
                               int64_t widthB) {
    int64_t i = threadIdx.x + blockDim.x * blockIdx.x;
    int64_t j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < widthB && j < heightA) {
        int64_t sum = 0;
        for (int64_t k = 0; k < widthA; k++) {
            int64_t a = M[j * widthA + k];
            int64_t b = N[k * widthB + i];
            sum += a * b;
        }
        P[j * widthB + i] = sum;
    }
}

int Row_A = 0, Col_A = 0, Row_B = 0, Col_B = 0;
int main(int argc, char **argv) {
    while (1) {
        char *fileName = new char[100];
        // 读取本进程所代表的chiplet编号
        int srcX = atoi(argv[1]);
        int srcY = atoi(argv[2]);
        int64_t *size_A = new int64_t[2];
        int64_t *size_B = new int64_t[2];
        int64_t *flag = new int64_t[1];
        int64_t *Size_A, *Size_B, *Flag;
        hipMalloc((void **)&Size_A, sizeof(int64_t) * 2);
        hipMalloc((void **)&Size_B, sizeof(int64_t) * 2);
        hipMalloc((void **)&Flag, sizeof(int64_t) * 1);

        receiveMessage(srcX, srcY, 0, 0, Flag, sizeof(int64_t) * 1);
        hipMemcpy(flag, Flag, sizeof(int64_t) * 1, hipMemcpyDeviceToHost);
        std::cout << "接收flag" << std::endl;
        if(flag[0] == 0)
            {
                std::cout << "结束" << std::endl;
                return 0;
            }
        else {
                std::cout << "正在运行" << std::endl;
                std::cout << "flag为" << flag[0] << std::endl;
        }
        receiveMessage(srcX, srcY, 0, 0, Size_A, sizeof(int64_t) * 2);
        std::cout << "接收Size_A" << std::endl;
        receiveMessage(srcX, srcY, 0, 0, Size_B, sizeof(int64_t) * 2);
        std::cout << "接收Size_B" << std::endl;

        hipMemcpy(size_A, Size_A, sizeof(int64_t) * 2, hipMemcpyDeviceToHost);
        hipMemcpy(size_B, Size_B, sizeof(int64_t) * 2, hipMemcpyDeviceToHost);
        Row_A = size_A[0];
        Col_A = size_A[1];
        Row_B = size_B[0];
        Col_B = size_B[1];
        int64_t *C = (int64_t *)malloc(sizeof(int64_t) * Col_B * Row_A);
        int64_t *A = (int64_t *)malloc(sizeof(int64_t) * Row_A * Col_A);

        int64_t *d_dataA, *d_dataB, *d_dataC;
        hipMalloc((void **)&d_dataA, sizeof(int64_t) * Row_A * Col_A);
        hipMalloc((void **)&d_dataB, sizeof(int64_t) * Row_B * Col_B);
        hipMalloc((void **)&d_dataC, sizeof(int64_t) * Col_B * Row_A);

        receiveMessage(srcX, srcY, 0, 0, d_dataA, Col_A * Row_A * sizeof(int64_t));
        std::cout << "接收d_dataA" << std::endl;
        receiveMessage(srcX, srcY, 0, 0, d_dataB, Col_B * Row_B * sizeof(int64_t));
        std::cout << "接收d_dataB" << std::endl;

        hipMemcpy(A, d_dataA, sizeof(int64_t) * Col_A * Row_A, hipMemcpyDeviceToHost);
        for (int64_t i = 0; i < Row_A * Col_A; i++) {
            std::cout << A[i];
            if (i % Col_A == 0 && i != 0)
                std::cout << std::endl;
            else
                std::cout << " ";
        }
        // calculate
        dim3 threadPerBlock(BLOCK_DIM, BLOCK_DIM);
        // dim3 blockNumber(1);
        dim3 blockNumber((Col_B + threadPerBlock.x - 1) / threadPerBlock.x,
                         (Row_A + threadPerBlock.y - 1) / threadPerBlock.y);
        matrix_mul_gpu<<<blockNumber, threadPerBlock>>>(d_dataA, d_dataB, d_dataC, Col_A, Row_A,
                                                        Col_B);
        hipMemcpy(C, d_dataC, sizeof(int64_t) * Row_A * Col_B, hipMemcpyDeviceToHost);
        for (int64_t i = 0; i < Row_A * Col_B; i++) {
            std::cout << C[i];
            if (i % Col_B == 0 && i != 0)
                std::cout << std::endl;
            else
                std::cout << " ";
        }
        sendMessage(0, 0, srcX, srcY, d_dataC, Row_A * Col_B * sizeof(int64_t));
        std::cout << "发送d_dataC" << std::endl;
        hipFree(d_dataA);
        hipFree(d_dataB);
        hipFree(d_dataC);
    }
    return 0;
}